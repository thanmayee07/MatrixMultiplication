#include "hip/hip_runtime.h"
%%cuda

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 256
#define BLOCK_SIZE 16

__global__ void matrixMul(float *a, float *b, float *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main() {
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
    int size = N * N * sizeof(float);

    // Allocate host memory
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);

    // Initialize host matrices a and b
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            a[i * N + j] = i + j;
            b[i * N + j] = i - j;
        }
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy host matrices to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Set grid and block dimensions
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Calculate the total number of floating-point operations
    long long total_flops = 2 * (long long)N * (long long)N * (long long)N;

    // Convert time to seconds
    double seconds = milliseconds / 1000.0;

    // Calculate GFLOPS
    double gflops = total_flops / (seconds * 1e9); // Convert time to seconds and GFLOPS to 1e9 scale

    printf("Time taken: %f milliseconds\n", milliseconds);
    printf("GFLOPS: %f\n", gflops);

    // Copy result from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
